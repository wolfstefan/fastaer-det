#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "kernel.h"
#include "../bboxUtils.h"
#include "../plugin_internal.h"

__global__ void rotBboxToPoly(
        int const numBboxes,
        float const * rotBboxes,
        float * polys)
{
    for (int index = blockIdx.x * blockDim.x + threadIdx.x;
         index < numBboxes;
         index += blockDim.x * gridDim.x)
    {
        float x = rotBboxes[index * 5 + 0];
        float y = rotBboxes[index * 5 + 1];
        float w = rotBboxes[index * 5 + 2] - 1;
        float h = rotBboxes[index * 5 + 3] - 1;
        float a = rotBboxes[index * 5 + 4];

        float w_h = w / 2.0f;
        float h_h = h / 2.0f;

        float cos = cosf(a);
        float sin = sinf(a);

        float x1 = x + cos * w_h - sin * -h_h;
        float x2 = x + cos * w_h - sin * h_h;
        float x3 = x + cos * -w_h - sin * h_h;
        float x4 = x + cos * -w_h - sin * -h_h;

        float y1 = y + sin * w_h + cos * -h_h;
        float y2 = y + sin * w_h + cos * h_h;
        float y3 = y + sin * -w_h + cos * h_h;
        float y4 = y + sin * -w_h + cos * -h_h;

        polys[index * 8 + 0] = x1;
        polys[index * 8 + 1] = y1;
        polys[index * 8 + 2] = x2;
        polys[index * 8 + 3] = y2;
        polys[index * 8 + 4] = x3;
        polys[index * 8 + 5] = y3;
        polys[index * 8 + 6] = x4;
        polys[index * 8 + 7] = y4;
    }
}

pluginStatus_t detectionInferenceRotated(
    hipStream_t stream,
    const int N,
    const int C1,
    const int C2,
    const bool shareLocation,
    const int backgroundLabelId,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const float confidenceThreshold,
    const float nmsThreshold,
    const DataType DT_BBOX,
    const void* locData,
    const DataType DT_SCORE,
    const void* confData,
    void* keepCount,
    void* topDetections,
    void* workspace,
    bool isNormalized,
    bool confSigmoid)
{
    // Batch size * number bbox per sample * 5 = total number of bounding boxes * 5
    const int locCount = N * C1;
    /*
     * shareLocation
     * Bounding box are shared among all classes, i.e., a bounding box could be classified as any candidate class.
     * Otherwise
     * Bounding box are designed for specific classes, i.e., a bounding box could be classified as one certain class or not (binary classification).
     */
    const int numLocClasses = shareLocation ? 1 : numClasses;

    size_t bboxDataSize = detectionRotatedForwardBBoxDataSize(N, numPredsPerClass, DataType::kFLOAT);
    void* bboxDataRaw = workspace;

    const int BS = 512;
    const int GS = (numPredsPerClass + BS - 1) / BS;
    rotBboxToPoly<<<GS, BS, 0, stream>>>(numPredsPerClass, (float const *)locData, (float *)bboxDataRaw);
    CSC(hipGetLastError(), STATUS_FAILURE);

    pluginStatus_t status;

    /*
     * bboxDataRaw format:
     * [batch size, numPriors (per sample), numLocClasses, 8]
     */
    // float for now
    void* bboxData;
    size_t bboxPermuteSize = detectionForwardBBoxPermuteSize(shareLocation, N, C1, DataType::kFLOAT);
    void* bboxPermute = nextWorkspacePtr((int8_t*) bboxDataRaw, bboxDataSize);

    /*
     * After permutation, bboxData format:
     * [batch_size, numLocClasses, numPriors (per sample) (numPredsPerClass), 8]
     * This is equivalent to swapping axis
     */
    ASSERT(shareLocation);
    if (!shareLocation)
    {
        status = permuteData(stream,
                             locCount,
                             numLocClasses,
                             numPredsPerClass,
                             4,
                             DataType::kFLOAT,
                             false,
                             bboxDataRaw,
                             bboxPermute);
        ASSERT_FAILURE(status == STATUS_SUCCESS);
        bboxData = bboxPermute;
    }
    /*
     * If shareLocation, numLocClasses = 1
     * No need to permute data on linear memory
     */
    else
    {
        bboxData = bboxDataRaw;
    }
    /*
     * Conf data format
     * [batch size, numPriors * param.numClasses, 1, 1]
     */
    const int numScores = N * C2;
    size_t scoresSize = detectionForwardPreNMSSize(N, C2);
    void* scores = nextWorkspacePtr((int8_t*) bboxPermute, bboxPermuteSize);
    // need a conf_scores
    /*
     * After permutation, bboxData format:
     * [batch_size, numClasses, numPredsPerClass, 1]
     */
    status = permuteData(stream,
                         numScores,
                         numClasses,
                         numPredsPerClass,
                         1,
                         DataType::kFLOAT,
                         confSigmoid,
                         confData,
                         scores);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    size_t indicesSize = detectionForwardPreNMSSize(N, C2);
    void* indices = nextWorkspacePtr((int8_t*) scores, scoresSize);

    size_t postNMSScoresSize = detectionForwardPostNMSSize(N, numClasses, topK);
    size_t postNMSIndicesSize = detectionForwardPostNMSSize(N, numClasses, topK);
    void* postNMSScores = nextWorkspacePtr((int8_t*) indices, indicesSize);
    void* postNMSIndices = nextWorkspacePtr((int8_t*) postNMSScores, postNMSScoresSize);

    //size_t sortingWorkspaceSize = sortScoresPerClassWorkspaceSize(N, numClasses, numPredsPerClass, FLOAT32);
    void* sortingWorkspace = nextWorkspacePtr((int8_t*) postNMSIndices, postNMSIndicesSize);
    // Sort the scores so that the following NMS could be applied.
    status = sortScoresPerClass(stream,
                                N,
                                numClasses,
                                numPredsPerClass,
                                backgroundLabelId,
                                confidenceThreshold,
                                DataType::kFLOAT,
                                scores,
                                indices,
                                sortingWorkspace);
    ASSERT_FAILURE(status == STATUS_SUCCESS);
    
    // NMS
    status = allClassNMSRotated(stream,
                         N,
                         numClasses,
                         numPredsPerClass,
                         topK,
                         nmsThreshold,
                         shareLocation,
                         isNormalized,
                         DataType::kFLOAT,
                         DataType::kFLOAT,
                         bboxData,
                         scores,
                         indices,
                         postNMSScores,
                         postNMSIndices,
                         false);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    // Sort the bounding boxes after NMS using scores
    status = sortScoresPerImage(stream,
                                N,
                                numClasses * topK,
                                DataType::kFLOAT,
                                postNMSScores,
                                postNMSIndices,
                                scores,
                                indices,
                                sortingWorkspace);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    // Gather data from the sorted bounding boxes after NMS
    status = gatherTopDetectionsRotated(stream,
                                 shareLocation,
                                 N,
                                 numPredsPerClass,
                                 numClasses,
                                 topK,
                                 keepTopK,
                                 DataType::kFLOAT,
                                 DataType::kFLOAT,
                                 indices,
                                 scores,
                                 bboxData,
                                 keepCount,
                                 topDetections);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    return STATUS_SUCCESS;
}

